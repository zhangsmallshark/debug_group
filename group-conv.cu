#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <string>

using namespace std;

#define B 128
#define R 3
#define S 3
#define TW 2
#define INTERNAL_TH 2
#define INTERNAL_TW 4
#define TH 2
#define TC 1 
#define H 28
#define W 28
#define C 128
#define N 128
#define TB 1

inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}

#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

class Conv{
public:
    unsigned int PAD;
    unsigned int hOut;
    unsigned int wOut;
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                    unsigned int pad,unsigned int r,unsigned int s,unsigned int stride);
    float *forward(float *input);
};

void Conv::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                      unsigned int pad,unsigned int r,unsigned int s,unsigned int stride){

    this->hOut = (H+2*pad - r)/stride + 1;
    this->wOut = (W+2*pad - s)/stride + 1;
    hipMalloc(&kernel,sizeof(float)*C*N*r*s);
    hipMalloc(&this->output,sizeof(float)*b*hOut*wOut*N);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/b,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/r,
            /*kernel_width=*/s);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/pad,
            /*pad_width=*/pad,
            /*vertical_stride=*/stride,
            /*horizontal_stride=*/stride,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/N,
            /*image_height=*/hOut,
            /*image_width=*/wOut);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = r*s*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,r*s*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}

float * Conv::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

__device__ void switch_function(int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(0-s)] += result;
				}
			}
		break;
		case 1:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(1-s)] += result;
				}
			}
		break;
		case 2:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(2-s)] += result;
				}
			}
		break;
		case 3:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(3-s)] += result;
				}
			}
		break;
		case 4:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(4-s)] += result;
				}
			}
		break;
		case 5:
			for ( int r = 0; r < 1; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(5-s)] += result;
				}
			}
		break;
		case 6:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(0-s)] += result;
				}
			}
		break;
		case 7:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(1-s)] += result;
				}
			}
		break;
		case 8:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(2-s)] += result;
				}
			}
		break;
		case 9:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(3-s)] += result;
				}
			}
		break;
		case 10:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(4-s)] += result;
				}
			}
		break;
		case 11:
			for ( int r = 0; r < 2; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(5-s)] += result;
				}
			}
		break;
		case 12:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(0-s)] += result;
				}
			}
		break;
		case 13:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(1-s)] += result;
				}
			}
		break;
		case 14:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(2-s)] += result;
				}
			}
		break;
		case 15:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(3-s)] += result;
				}
			}
		break;
		case 16:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(4-s)] += result;
				}
			}
		break;
		case 17:
			for ( int r = 1; r < 3; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(5-s)] += result;
				}
			}
		break;
		case 18:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(0-s)] += result;
				}
			}
		break;
		case 19:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(1-s)] += result;
				}
			}
		break;
		case 20:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(2-s)] += result;
				}
			}
		break;
		case 21:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(3-s)] += result;
				}
			}
		break;
		case 22:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(4-s)] += result;
				}
			}
		break;
		case 23:
			for ( int r = 2; r < 3; r++) {
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(5-s)] += result;
				}
			}
		break;

	}
}

__device__ void load_input(float * input, float * shared_input, int h_start, int h_end,
                           int c_id, int h_offset, int batch_start, int *channels){
    unsigned int h_len = h_end - h_start;
    for(int i=threadIdx.x;i<h_len*W*TB;i+=blockDim.x){
        int batch_id = batch_start + i/(h_len*W);
        int c = channels[c_id];
        int hw_id = i % (h_len*W);
        int h = hw_id/W;
        int w = hw_id%W;
        int local_h = h+h_offset;
        int local_w = w+1;
        int local_h_no_padding = h_start+h;
        int local_w_no_padding = w;
        shared_input[(batch_id - batch_start)*(TH+2)*(W+2) + local_h*(W+2)+local_w]
                = input[c*(H)*(W)+local_h_no_padding*(W)+local_w_no_padding];
    }
}
__global__ void conv2d_no_padding(float * __restrict__ input,const float * __restrict__ kernel,
                                  float * __restrict__ outputs, int *channels, int *filters, int *channel_ptr,
                                  int *filter_ptr){
    __shared__ float input_tile[(TH+2)*(W+2)*TB];
    __shared__ float shared_kernel[9*N];
    int THS = (H - 1)/TH + 1;
    int TBS = (B - 1)/TB + 1;
    int group_id = blockIdx.x / (TBS*THS);
    int num_channels = channel_ptr[group_id+1] - channel_ptr[group_id];
    int num_filters = filter_ptr[group_id+1] - filter_ptr[group_id];

    int blk_id = blockIdx.x % (TBS*THS);
    int batch_start = (blk_id / THS)*TB;
    int h_start =  (blk_id % THS) * TH;
    float local_compute[INTERNAL_TH*INTERNAL_TW] = {0.0f};
    int block_th = min(TH,(H - h_start));
    int inner_ths = (block_th - 1)/INTERNAL_TH + 1;
    int inner_tws = (W - 1)/INTERNAL_TW + 1;
    int h_copy_start = max(h_start - 1, 0);
    int h_copy_end = min(h_start + block_th + 1, H);
    int h_offset = ((h_start - 1) < 0)?1:0;
    for(unsigned int i=threadIdx.x;i<(TH+2)*(W+2)*TB;i+=blockDim.x){
        input_tile[i] = 0.0f;
    }
    for(int i = threadIdx.x; i<9*N; i+=blockDim.x){
        shared_kernel[i] = 1.0f;
    }
    __syncthreads();
    for(int c=0;c<num_channels;c++){
        int channel_index = channels[channel_ptr[group_id] + c];
        load_input(input,input_tile,h_copy_start,h_copy_end,channel_index,h_offset,batch_start,channels);
        __syncthreads();
        for(int i=threadIdx.x;i<inner_ths*inner_tws*TB;i+=blockDim.x) {
            int b = i / (inner_ths * inner_tws);
            int batch_id = b + batch_start;
            int local_h = ((i % (inner_ths * inner_tws)) / inner_tws) * INTERNAL_TH;
            int local_w = ((i % (inner_ths * inner_tws)) % inner_tws) * INTERNAL_TW;
            int h_end = min(block_th - local_h + 2, INTERNAL_TH + 2);
            int w_end = min(W - local_w + 2, INTERNAL_TW + 2);
            for(int n=0;n<num_filters;++n){
                int output_channel_index = filters[filter_ptr[group_id]+n];
                float *local_kernel = &shared_kernel[0];
                for(int h=0;h<h_end;++h){
                    for(int w=0;w<w_end;++w){
                        float v = input_tile[b*(TH+2)*(W+2)+(local_h+h)*(W+2)+local_w+w];
                        int linear_id = h*(INTERNAL_TW+2) + w;
                        switch_function(linear_id,local_kernel,v,local_compute);
                    }
                }
                for(int h=0;h<INTERNAL_TH;h++){
                    for(int w=0;w<INTERNAL_TW;++w){
                        int h_out = h_start + h + local_h;
                        int w_out = w + local_w;
                        if(h_out>=h_start+block_th||w_out>=W){
                            continue;
                        }
                        atomicAdd(&outputs[batch_id*N*H*W+output_channel_index*H*W+h_out*W+w_out],
                                  local_compute[h*INTERNAL_TW+w]);
                    }
                }
                for(int j=0;j<INTERNAL_TW*INTERNAL_TH;++j){
                    local_compute[j] = 0.0f;
                }
            }
        }
    }
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}

#define PTR_S 5
#define C_S 328
#define F_S 128

int main(int argc, char *argv[]){
    int groups = 4;
    int filters_ptr[5] = {0, 32, 64, 96, 128};
    int filters[128] = {0, 5, 9, 13, 14, 17, 20, 21, 23, 25, 26, 30, 34, 36, 39, 51, 57, 65, 70, 84, 89, 96, 97, 98, 99, 101, 109, 110, 118, 119, 120, 124, 1, 2, 8, 12, 15, 27, 31, 34, 35, 41, 46, 50, 52, 53, 54, 56, 69, 73, 80, 81, 82, 90, 92, 95, 104, 105, 107, 108, 114, 115, 123, 126, 3, 6, 7, 22, 28, 29, 32, 38, 44, 55, 58, 64, 66, 67, 68, 74, 75, 76, 79, 83, 85, 86, 88, 91, 94, 102, 103, 106, 113, 116, 122, 125, 4, 10, 11, 16, 18, 19, 24, 33, 40, 42, 43, 45, 47, 48, 49, 59, 60, 61, 62, 63, 71, 72, 77, 78, 87, 93, 100, 111, 112, 117, 121, 127};
    int channels_ptr[5] = {0, 71, 146, 239, 328};
    int channels[328] = {0, 1, 2, 5, 6, 10, 11, 12, 14, 17, 20, 22, 23, 24, 25, 26, 30, 33, 35, 36, 37, 38, 39, 40, 41, 43, 44, 48, 52, 57, 58, 59, 61, 62, 63, 64, 65, 68, 69, 71, 72, 74, 75, 76, 78, 79, 80, 85, 86, 87, 88, 90, 91, 92, 95, 96, 98, 99, 100, 101, 103, 107, 110, 111, 113, 114, 115, 117, 119, 120, 121, 1, 3, 4, 5, 6, 7, 9, 12, 13, 14, 15, 22, 27, 28, 29, 31, 33, 35, 36, 37, 40, 42, 43, 46, 48, 49, 51, 52, 53, 55, 58, 59, 61, 64, 65, 66, 67, 68, 72, 73, 74, 76, 77, 78, 79, 81, 82, 83, 88, 89, 90, 94, 97, 98, 99, 101, 102, 103, 104, 105, 107, 108, 110, 111, 112, 113, 117, 118, 119, 120, 123, 124, 125, 126, 127, 3, 5, 6, 7, 8, 10, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 26, 28, 29, 30, 31, 33, 35, 38, 39, 41, 42, 44, 45, 46, 47, 48, 49, 51, 53, 54, 55, 58, 59, 60, 61, 62, 63, 64, 66, 67, 69, 70, 71, 73, 74, 76, 77, 78, 81, 82, 83, 84, 85, 86, 87, 88, 91, 92, 93, 94, 96, 97, 98, 99, 100, 101, 102, 103, 104, 105, 106, 107, 108, 109, 110, 112, 115, 116, 118, 119, 122, 123, 124, 125, 126, 127, 0, 1, 3, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31, 32, 34, 35, 36, 37, 38, 41, 42, 45, 46, 49, 50, 51, 52, 54, 60, 61, 62, 64, 65, 66, 67, 70, 71, 75, 76, 78, 79, 80, 81, 83, 84, 85, 86, 87, 88, 91, 93, 94, 95, 97, 98, 99, 100, 101, 102, 104, 105, 106, 108, 109, 110, 111, 112, 114, 115, 117, 120, 121, 122, 125, 126, 127};

    int *device_channels;
    int *device_filters;
    int *device_channels_ptr;
    int *device_filters_ptr;

    hipMalloc(&device_filters,F_S*sizeof(int));
    hipMalloc(&device_channels,C_S*sizeof(int));
    hipMalloc(&device_channels_ptr,PTR_S*sizeof(int));
    hipMalloc(&device_filters_ptr,PTR_S*sizeof(int));

    hipMemcpy(device_filters,filters,F_S*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(device_channels,channels,C_S*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(device_channels_ptr,channels_ptr,PTR_S*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(device_filters_ptr,filters_ptr,PTR_S*sizeof(int),hipMemcpyHostToDevice);
    
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    float *input = new float[B*C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<B*C*H*W;++i){
        input[i] = 1.0f;
    }
    float *device_input;
    hipMalloc(&device_input,B*C*H*W*sizeof(float));
    hipMemcpy(device_input,input,B*C*H*W*sizeof(float),hipMemcpyHostToDevice);

    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 0.0f;
    }
    float *device_k;
    hipMalloc(&device_k,C*N*9*sizeof(float));
    float *out;
    hipMalloc(&out,B*N*H*W*sizeof(float));

    for(int i=0;i<groups;++i){
        for(int j = filters_ptr[i];j<filters_ptr[i+1];j++){
            int filter = filters[j];
            for(int k = channels_ptr[i];k<channels_ptr[i+1];k++){
                int channel = channels[k];
                for(int r=0;r<3;++r){
                    for(int s=0;s<3;++s){
                        K[filter*C*9+channel*9+r*3+s] = 1.0f;
                    }
                }
            }
        }
    }
    Conv conv;
    conv.initialize(B,C,H,W,N,1,3,3,1);
    hipMemcpy(conv.kernel,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);
    float *out_cudnn = conv.forward(device_input);
    float *out_cudnn_host = new float[N*H*W*B];
    hipMemcpy(out_cudnn_host,out_cudnn,B*N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    unsigned int gridDim = ((H - 1)/TH + 1)*((B - 1)/TB + 1);
    unsigned int bdim = ((TH - 1)/INTERNAL_TH + 1)*((W - 1)/INTERNAL_TW + 1)*TB;
    hipEventRecord(event_start);
    hipMemset(out,0,N*H*W*sizeof(float));
    conv2d_no_padding<<<gridDim*groups,bdim>>>(device_input,device_k,out,device_channels,
                                               device_filters,device_channels_ptr,device_filters_ptr);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    chkerr(hipDeviceSynchronize());
    chkerr(hipGetLastError());
    float *out_tdc = new float[B*N*H*W];
    hipMemcpy(out_tdc,out,B*N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    hipEventRecord(event_start);
    out_cudnn = conv.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_cudnn;
    hipEventElapsedTime(&time_cudnn, event_start, event_stop);
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<TH<<","<<INTERNAL_TH<<","<<INTERNAL_TW<<","<<
        time_cudnn<<","<<time_tdc<<","<< check_diff(out_cudnn_host,out_tdc,B*N*H*W)<<endl;

    string out_file = "res0.txt";
    ofstream out_s;
    if (time_tdc < time_cudnn) {
        out_s.open(out_file, ios::binary | ios::app | ios::in | ios::out);
        out_s<<C<<","<<N<<","<<H<<","<<W<<","<<TH<<","<<INTERNAL_TH<<","<<INTERNAL_TW<<","<<time_cudnn<<","<<time_tdc<<","<< check_diff(out_cudnn_host,out_tdc,B*N*H*W)<<"\n";
        out_s.close();
        cout << "Find it " << endl;
    }

    return 0;
}